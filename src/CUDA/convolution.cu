#include "hip/hip_runtime.h"
#include "convolution.cuh"

#define BLOCK_SIZE 32 // sqrt of max number of threads per block
using namespace std;

__global__ void convolve2d_helper(float* image, int row_size, int col_size, float* kernel, int kernel_size, int kernel_offset, float* result){

    // calculate row and column positions
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    // check bounds
    if(row >= row_size || col >= col_size) return;

    int iFlip, jFlip; // flipped kernel indices
    int ii, jj;
    float temp = 0;

    for(int i = 0; i < kernel_size; ++i){

        iFlip = kernel_size - 1 - i;

        for(int j = 0; j < kernel_size; ++j){

            jFlip = kernel_size - 1 - j;

            ii = row + (kernel_offset - iFlip);
            jj = col + (kernel_offset - jFlip);

            if(ii >= 0 && ii < row_size && jj >= 0 && jj < col_size) {
                temp += image[ii * col_size + jj] * kernel[iFlip * kernel_size + jFlip];
            }
        }
    }

    result[row * col_size + col] = temp;
}


__global__ void convolve2d_helper_opt(float* image, int row_size, int col_size, const float* __restrict__ kernel, int kernel_size, int kernel_offset, float* result, int tile_width){
    
    // calculate row and column positions
    int row = blockIdx.y * tile_width + threadIdx.y;
    int col = blockIdx.x * tile_width + threadIdx.x;

    int m_row = row - kernel_offset;
    int m_col = col - kernel_offset;

    __shared__ float tile[BLOCK_SIZE * BLOCK_SIZE];

    if (m_row >= 0 && m_row < row_size && m_col >=0 && m_col < col_size) {
        tile[threadIdx.y * BLOCK_SIZE + threadIdx.x] = image[m_row * col_size + m_col];
    } else {
        tile[threadIdx.y * BLOCK_SIZE + threadIdx.x] = 0;
    }

    __syncthreads();

    float temp = 0;
    if (threadIdx.y < tile_width && threadIdx.x < tile_width && row < row_size  && col < col_size) {

        for (int i = 0; i < kernel_size; ++i) {
            for (int j = 0; j < kernel_size; ++j) {
                temp += kernel[i * kernel_size + j] * tile[(threadIdx.y + i) * BLOCK_SIZE + (threadIdx.x + j)];
            }
        }

        result[row * col_size + col] = temp;
    }
}


namespace convolution {

    vector<vector<float>> tile_convolve2d(vector<vector<float>>& image, vector<vector<float>>& kernel){
        int paddingSize = -1;
        int topPadding, bottomPadding;
        unsigned long long image_size [2] = {image.size(), image[0].size()};

        // error checking
        if(paddingSize < -1) throw invalid_argument("Padding size must be a non negative number.");

        // check if kernel is empty
        if(kernel.size() == 0 && kernel[0].size() == 0) throw invalid_argument("kernel must be non empty.");

        // check if image is empty
        if(image.size() == 0 && image[0].size() == 0) throw invalid_argument("image must be non empty.");

        // ensure every row is the same length
        for(int i = 1; i < image.size(); ++i){
            if(image[i].size() != image_size[1]) throw invalid_argument("image must be a rectangular matrix.");
        }

        // ensure every row is the same length
        int kernel_size = kernel[0].size();
        for(int i = 1; i < kernel.size(); ++i){
            if(kernel[i].size() != kernel_size) throw invalid_argument("kernel must be a rectangular matrix.");
        }

        // end error checking

        // calculate padding size
        if(paddingSize == -1) {

            topPadding = ceil((kernel.size()-1) / 2.);
            bottomPadding = floor((kernel[0].size()-1) / 2.);

        } else {
            topPadding = paddingSize;
            bottomPadding = paddingSize;
        }

        // calculate new size of image based on padding size
        int row_size = (topPadding + bottomPadding + image_size[0]);
        int col_size = (topPadding + bottomPadding + image_size[1]);
        int size = row_size * col_size;

        // allocate space for linear indexed arrays
        float* linear_image = (float*)malloc(size * sizeof(float));
        float* result = (float*)malloc(size * sizeof(float));
        float* linear_kernel = (float*)malloc(kernel.size()*kernel[0].size() * sizeof(float));

        auto linear_start = chrono::high_resolution_clock::now();

        int index;
        // create linear indexed image with padding
        for (int i = 0; i < row_size; ++i) {
            for (int j = 0; j < col_size; ++j) {
                index = i*col_size + j;
                if (j < bottomPadding || j >= bottomPadding + image_size[1]) {
                    linear_image[index] = 0;
                } else if (i < bottomPadding || i >= bottomPadding + image_size[0]) {
                    linear_image[index] = 0;
                } else {
                    linear_image[index] = image[i-bottomPadding][j-bottomPadding];
                }
            }
        }

        // create linear indexed filter
        for (int i = 0; i < kernel.size(); ++i) {
            for (int j = 0; j < kernel[0].size(); ++j) {
                linear_kernel[i*kernel.size() + j] = kernel[kernel.size() - 1 - i][kernel.size() - 1 - j];
            }
        }

        auto linear_end = chrono::high_resolution_clock::now();
        chrono::duration<double, std::milli> linear_ms = linear_end - linear_start;
        cout << "time taken to flatten array: " << linear_ms.count() << endl;

        float* d_image;
        float* d_result;
        float* d_kernel;

        // allocate vectors for GPU
        hipMalloc(&d_image, size*sizeof(float));
        
        hipMalloc(&d_result, size*sizeof(float));
        hipMalloc(&d_kernel, kernel.size()*kernel.size()*sizeof(float));
        
        // copy data to GPU
        hipMemcpy(d_image, linear_image, size*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_kernel, linear_kernel, kernel.size()*kernel.size()*sizeof(float), hipMemcpyHostToDevice);


        int tile_width = BLOCK_SIZE - (kernel.size() -1);
        
        dim3 num_blocks(ceil(col_size / (float) tile_width), ceil(row_size / (float) tile_width), 1);
        dim3 num_threads(BLOCK_SIZE, BLOCK_SIZE, 1);

        printf("Grid : {%d, %d, %d} blocks. Blocks : {%d, %d, %d} threads.\n",
        num_blocks.x, num_blocks.y, num_blocks.z, num_threads.x, num_threads.y, num_threads.z);

        int offset = kernel.size() / 2.; // center of filter
        
        // call kernel
        auto start = chrono::high_resolution_clock::now();
        //convolve2d_helper_opt<<<num_blocks, num_threads>>>(d_image, row_size, col_size, d_kernel, kernel.size(), offset, d_result, tile_width);
        convolve2d_helper_opt<<<num_blocks, num_threads>>>(d_image, row_size, col_size, d_kernel, kernel.size(), offset, d_result, tile_width);
        hipDeviceSynchronize();

        hipError_t err = hipGetLastError();

        if ( err != hipSuccess )
        {
            printf("CUDA Error: %s\n", hipGetErrorString(err));       
        }

        auto end = chrono::high_resolution_clock::now();
        chrono::duration<double, std::milli> time  = end - start;
        cout << "time in kernel: " << time.count() << endl;

        // copy results from device to host
        hipMemcpy(result, d_result, size*sizeof(float), hipMemcpyDeviceToHost);
        
        // initialize return vector
        vector<vector<float>> out( row_size , vector<float> (col_size, 0));
        
        // add results to 2d vector to return
        for (int i = 0; i < row_size; ++i) {
            for (int j = 0; j < col_size; ++j) {
                out[i][j] = result[i*col_size + j];
            }
        }

        // free host memory
        free(linear_image);
        free(result);
        free(linear_kernel);

        // free device memory
        hipFree(d_image);
        hipFree(d_result);
        hipFree(d_kernel);

        return out;
    }

    vector<vector<float>> convolve2d(vector<vector<float>>& image, vector<vector<float>>& kernel, int threads, int paddingSize){

        int topPadding, bottomPadding;
        unsigned long long image_size [2] = {image.size(), image[0].size()};

        // error checking
        if(paddingSize < -1) throw invalid_argument("Padding size must be a non negative number.");

        // check if kernel is empty
        if(kernel.size() == 0 && kernel[0].size() == 0) throw invalid_argument("kernel must be non empty.");

        // check if image is empty
        if(image.size() == 0 && image[0].size() == 0) throw invalid_argument("image must be non empty.");

        // ensure every row is the same length
        for(int i = 1; i < image.size(); ++i){
            if(image[i].size() != image_size[1]) throw invalid_argument("image must be a rectangular matrix.");
        }

        // ensure every row is the same length
        int kernel_size = kernel[0].size();
        for(int i = 1; i < kernel.size(); ++i){
            if(kernel[i].size() != kernel_size) throw invalid_argument("kernel must be a rectangular matrix.");
        }

        // end error checking

        // calculate padding size
        if(paddingSize == -1) {

            topPadding = ceil((kernel.size()-1) / 2.);
            bottomPadding = floor((kernel[0].size()-1) / 2.);

        } else {
            topPadding = paddingSize;
            bottomPadding = paddingSize;
        }

        // calculate new size of image based on padding size
        int row_size = (topPadding + bottomPadding + image_size[0]);
        int col_size = (topPadding + bottomPadding + image_size[1]);
        int size = row_size * col_size;

        // allocate space for linear indexed arrays
        float* linear_image = (float*)malloc(size * sizeof(float));
        float* result = (float*)malloc(size * sizeof(float));
        float* linear_kernel = (float*)malloc(kernel.size()*kernel[0].size() * sizeof(float));

        auto linear_start = chrono::high_resolution_clock::now();

        int index;
        // create linear indexed image with padding
        for (int i = 0; i < row_size; ++i) {
            for (int j = 0; j < col_size; ++j) {
                index = i*col_size + j;
                if (j < topPadding || j >= topPadding + image_size[1]) {
                    linear_image[index] = 0;
                } else if (i < topPadding || i >= topPadding + image_size[0]) {
                    linear_image[index] = 0;
                } else {
                    linear_image[index] = image[i-topPadding][j-topPadding];
                }
            }
        }

        // create linear indexed filter
        for (int i = 0; i < kernel.size(); ++i) {
            for (int j = 0; j < kernel[0].size(); ++j) {
                linear_kernel[i*kernel.size() + j] = kernel[i][j];
            }
        }

        auto linear_end = chrono::high_resolution_clock::now();
        chrono::duration<double, std::milli> linear_ms = linear_end - linear_start;
        cout << "time taken to flatten array: " << linear_ms.count() << endl;

        float* d_image;
        float* d_result;
        float* d_kernel;

        // allocate vectors for GPU
        hipMalloc(&d_image, size*sizeof(float));
        hipMalloc(&d_result, size*sizeof(float));
        hipMalloc(&d_kernel, kernel.size()*kernel.size()*sizeof(float));
        
        // copy data to GPU
        hipMemcpy(d_image, linear_image, size*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_kernel, linear_kernel, kernel.size()*kernel.size()*sizeof(float), hipMemcpyHostToDevice);

        cout << "threads: " << threads << endl;
        int X_THREADS = threads;
        int Y_THREADS = X_THREADS;
        int X_BLOCKS = (col_size + X_THREADS - 1) / X_THREADS;
        int Y_BLOCKS = (row_size + Y_THREADS - 1) / Y_THREADS;

        dim3 block_dim(X_THREADS, Y_THREADS);
        dim3 grid_dim(X_BLOCKS, Y_BLOCKS);

        int offset = kernel.size() / 2.; // center of filter
        
        // call kernel
        auto start = chrono::high_resolution_clock::now();
        convolve2d_helper<<<grid_dim, block_dim>>>(d_image, row_size, col_size, d_kernel, kernel.size(), offset, d_result);
        hipDeviceSynchronize();

        hipError_t err = hipGetLastError();

        if ( err != hipSuccess )
        {
            printf("CUDA Error: %s\n", hipGetErrorString(err));       
        }

        auto end = chrono::high_resolution_clock::now();
        chrono::duration<double, std::milli> time  = end - start;
        cout << "time in kernel: " << time.count() << endl;

        // copy results from device to host
        hipMemcpy(result, d_result, size*sizeof(float), hipMemcpyDeviceToHost);
        
        // initialize return vector
        vector<vector<float>> out( row_size , vector<float> (col_size, 0));
        
        // add results to 2d vector to return
        for (int i = 0; i < row_size; ++i) {
            for (int j = 0; j < col_size; ++j) {
                out[i][j] = result[i*col_size + j];
            }
        }

        // free host memory
        free(linear_image);
        free(result);
        free(linear_kernel);

        // free device memory
        hipFree(d_image);
        hipFree(d_result);
        hipFree(d_kernel);

        return out;
    }
}
